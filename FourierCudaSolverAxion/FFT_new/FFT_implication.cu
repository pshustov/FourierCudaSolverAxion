#include "hip/hip_runtime.h"
#include "stdafx.h"

__global__ void kernelNorm(const int N, double* V)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N)
	{
		V[i] = V[i] / N;
	}
}
__global__ void kernelNorm(const int N, complex* V)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N)
	{
		V[i] = V[i] / N;
	}
}

__global__ void kernelForwardNorm(const int size, const int N, const double L, double *V)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		V[i] = V[i] * L / N;
	}
}
__global__ void kernelForwardNorm(const int size, const int N, const double L, complex *V)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		V[i] = V[i] * L / N;
	}
}
__global__ void kernelInverseNorm(const int size, const int N, const double L, double* V)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		V[i] = V[i] / L;
	}
}
__global__ void kernelInverseNorm(const int size, const int N, const double L, complex* V)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		V[i] = V[i] / L;
	}
}

void cufft(cudaCVector &f, cudaCVector &F)
{
	int NX = (int)f.get_N();
	auto BATCH = 1;

	hipfftHandle plan;
	if (hipfftPlan1d(&plan, NX, HIPFFT_Z2Z, BATCH) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: Plan creation failed");
		return;
	}
	if (hipfftExecZ2Z(plan, (hipfftDoubleComplex*)f.get_Array(), (hipfftDoubleComplex*)F.get_Array(), HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: ExecZ2Z Forward failed");
		return;
	}
	hipfftDestroy(plan);
	hipDeviceSynchronize();
}
void cufft(cudaRVector &f, cudaCVector &F)
{
	int NX = (int)f.get_N();
	auto BATCH = 1;

	hipfftHandle plan;
	if (hipfftPlan1d(&plan, NX, HIPFFT_D2Z, BATCH) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: Plan creation failed");
		return;
	}
	if (hipfftExecD2Z(plan, (hipfftDoubleReal*)f.get_Array(), (hipfftDoubleComplex*)F.get_Array()) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: ExecD2Z Forward failed");
		return;
	}
	hipfftDestroy(plan);
	hipDeviceSynchronize();
}
void cufft(cudaCVector3 &f, cudaCVector3 &F)
{
	int NX = (int)f.get_N1();
	int NY = (int)f.get_N2();
	int NZ = (int)f.get_N3();

	hipfftHandle plan;
	if (hipfftPlan3d(&plan, NX, NY, NZ, HIPFFT_Z2Z) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: Plan creation failed");
		return;
	}
	if (hipfftExecZ2Z(plan, (hipfftDoubleComplex*)f.get_Array(), (hipfftDoubleComplex*)F.get_Array(), HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: 3D ExecZ2Z Forward failed");
		return;
	}
	hipfftDestroy(plan);
	hipDeviceSynchronize();
}
void cufft(cudaRVector3 &f, cudaCVector3 &F)
{
	int NX = (int)f.get_N1();
	int NY = (int)f.get_N2();
	int NZ = (int)f.get_N3();

	hipfftHandle plan;
	if (hipfftPlan3d(&plan, NX, NY, NZ, HIPFFT_D2Z) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: Plan creation failed");
		return;
	}
	if (hipfftExecD2Z(plan, (hipfftDoubleReal*)f.get_Array(), (hipfftDoubleComplex*)F.get_Array()) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: 3D ExecD2Z Forward failed");
		return;
	}
	hipfftDestroy(plan);
	hipDeviceSynchronize();
}

void cuifft(cudaCVector &F, cudaCVector &f)
{
	int NX = (int)F.get_N();
	int BATCH = 1;

	hipfftHandle plan;
	if (hipfftPlan1d(&plan, NX, HIPFFT_Z2Z, BATCH) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: Plan creation failed");
		return;
	}
	if (hipfftExecZ2Z(plan, (hipfftDoubleComplex*)F.get_Array(), (hipfftDoubleComplex*)f.get_Array(), HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: ExecZ2Z Inverce failed");
		return;
	}
	hipfftDestroy(plan);
	hipDeviceSynchronize();

	dim3 block(BLOCK_SIZE);
	dim3 grid((unsigned int)ceil((double)f.get_N() / (double)BLOCK_SIZE));
	kernelNorm <<<grid, block>>> (NX, f.get_Array());
	hipDeviceSynchronize();
}
void cuifft(cudaCVector &F, cudaRVector &f)
{
	int NX = (int)F.get_N();
	int BATCH = 1;

	hipfftHandle plan;
	if (hipfftPlan1d(&plan, NX, HIPFFT_Z2D, BATCH) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: Plan creation failed");
		return;
	}
	if (hipfftExecZ2D(plan, (hipfftDoubleComplex*)F.get_Array(), (hipfftDoubleReal*)f.get_Array()) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: ExecZ2Z Inverce failed");
		return;
	}
	hipfftDestroy(plan);
	hipDeviceSynchronize();

	dim3 block(BLOCK_SIZE);
	dim3 grid((unsigned int)ceil((double)NX / (double)BLOCK_SIZE));
	kernelNorm <<<grid, block>>> (NX, f.get_Array());
	hipDeviceSynchronize();
}
void cuifft(cudaCVector3 &F, cudaCVector3 &f)
{
	int NX = (int)F.get_N1();
	int NY = (int)F.get_N2();
	int NZ = (int)F.get_N3();

	hipfftHandle plan;
	if (hipfftPlan3d(&plan, NX, NY, NZ, HIPFFT_Z2Z) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: Plan creation failed");
		return;
	}
	if (hipfftExecZ2Z(plan, (hipfftDoubleComplex*)F.get_Array(), (hipfftDoubleComplex*)f.get_Array(), HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: ExecZ2Z Inverce failed");
		return;
	}
	hipfftDestroy(plan);
	hipDeviceSynchronize();

	dim3 block(BLOCK_SIZE);
	dim3 grid((unsigned int)ceil((double)NX*NY*NZ / (double)BLOCK_SIZE));
	kernelNorm<<<grid, block>>>(NX*NY*NZ, f.get_Array());
	hipDeviceSynchronize();

}
void cuifft(cudaCVector3 &F, cudaRVector3 &f)
{
	int NX = (int)F.get_N1();
	int NY = (int)F.get_N2();
	int NZ = (int)F.get_N3();

	hipfftHandle plan;
	if (hipfftPlan3d(&plan, NX, NY, NZ, HIPFFT_Z2D) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: Plan creation failed");
		return;
	}
	if (hipfftExecZ2D(plan, (hipfftDoubleComplex*)F.get_Array(), (hipfftDoubleReal*)f.get_Array()) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: ExecZ2Z Inverce failed");
		return;
	}
	hipfftDestroy(plan);
	hipDeviceSynchronize();

	dim3 block(BLOCK_SIZE);
	dim3 grid((unsigned int)ceil((double)NX*NY*NZ / (double)BLOCK_SIZE));
	kernelNorm<<<grid, block>>>(NX*NY*NZ, f.get_Array());
	hipDeviceSynchronize();
}

void cuFFT::forward(cudaCVector &f, cudaCVector &F)
{
	if (hipfftExecZ2Z(planZ2Z, (hipfftDoubleComplex*)f.get_Array(), (hipfftDoubleComplex*)F.get_Array(), HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: ExecZ2Z Forward failed");
		return;
	}
	hipDeviceSynchronize();
	
	dim3 block(BLOCK_SIZE);
	dim3 grid((unsigned int)ceil((double)F.get_N() / (double)BLOCK_SIZE));
	kernelForwardNorm<<<grid, block>>>(F.get_N(), N, L, F.get_Array());
	hipDeviceSynchronize();
}
void cuFFT::forward(cudaRVector &f, cudaCVector &F)
{
	if (hipfftExecD2Z(planD2Z, (hipfftDoubleReal*)f.get_Array(), (hipfftDoubleComplex*)F.get_Array()) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: ExecD2Z Forward failed");
		return;
	}
	hipDeviceSynchronize();
	
	dim3 block(BLOCK_SIZE);
	dim3 grid((unsigned int)ceil((double)F.get_N() / (double)BLOCK_SIZE));
	kernelForwardNorm<<<grid, block>>>(F.get_N(), N, L, F.get_Array());
	hipDeviceSynchronize();
}
void cuFFT::forward(cudaCVector3 &f, cudaCVector3 &F)
{
	if (hipfftExecZ2Z(planZ2Z, (hipfftDoubleComplex*)f.get_Array(), (hipfftDoubleComplex*)F.get_Array(), HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: 3D ExecZ2Z Forward failed");
		return;
	}
	hipDeviceSynchronize();
	
	dim3 block(BLOCK_SIZE);
	dim3 grid((unsigned int)ceil((double)F.size() / (double)BLOCK_SIZE));
	kernelForwardNorm<<<grid, block>>>(F.size(), N, L, F.get_Array());
	hipDeviceSynchronize();
}
void cuFFT::forward(cudaRVector3 &f, cudaCVector3 &F)
{
	if (hipfftExecD2Z(planD2Z, (hipfftDoubleReal*)f.get_Array(), (hipfftDoubleComplex*)F.get_Array()) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: 3D ExecD2Z Forward failed");
		return;
	}
	hipDeviceSynchronize();
	
	dim3 block(BLOCK_SIZE);
	dim3 grid((unsigned int)ceil((double)F.size() / (double)BLOCK_SIZE));
	kernelForwardNorm<<<grid, block>>>(F.size(), N, L, F.get_Array());
	hipDeviceSynchronize();
}

void cuFFT::inverce(cudaCVector &F, cudaCVector &f)
{
	if (hipfftExecZ2Z(planZ2Z, (hipfftDoubleComplex*)F.get_Array(), (hipfftDoubleComplex*)f.get_Array(), HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: ExecZ2Z Inverce failed");
		return;
	}
	hipDeviceSynchronize();

	dim3 block(BLOCK_SIZE);
	dim3 grid((unsigned int)ceil((double)f.get_N() / (double)BLOCK_SIZE));
	kernelInverseNorm<<<grid, block>>>(f.get_N(), N, L, f.get_Array());
	hipDeviceSynchronize();
}
void cuFFT::inverce(cudaCVector &F, cudaRVector &f)
{
	if (hipfftExecZ2D(planZ2D, (hipfftDoubleComplex*)F.get_Array(), (hipfftDoubleReal*)f.get_Array()) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: ExecZ2Z Inverce failed");
		return;
	}
	hipDeviceSynchronize();

	dim3 block(BLOCK_SIZE);
	dim3 grid((unsigned int)ceil((double)f.get_N() / (double)BLOCK_SIZE));
	kernelInverseNorm<<<grid, block>>>(f.get_N(), N, L, f.get_Array());
	hipDeviceSynchronize();
}
void cuFFT::inverce(cudaCVector3 &F, cudaCVector3 &f)
{
	if (hipfftExecZ2Z(planZ2Z, (hipfftDoubleComplex*)F.get_Array(), (hipfftDoubleComplex*)f.get_Array(), HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: 3D ExecZ2Z Inverce failed");
		return;
	}
	hipDeviceSynchronize();

	dim3 block(BLOCK_SIZE);
	dim3 grid((unsigned int)ceil((double)f.size() / (double)BLOCK_SIZE));
	kernelInverseNorm<<<grid, block>>>(f.size(), N, L, f.get_Array());
	hipDeviceSynchronize();
}
void cuFFT::inverce(cudaCVector3 &F, cudaRVector3 &f)
{
	if (hipfftExecZ2D(planZ2D, (hipfftDoubleComplex*)F.get_Array(), (hipfftDoubleReal*)f.get_Array()) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: 3D ExecZ2Z Inverce failed");
		return;
	}
	hipDeviceSynchronize();

	dim3 block(BLOCK_SIZE);
	dim3 grid((unsigned int)ceil((double)f.size() / (double)BLOCK_SIZE));
	kernelInverseNorm<<<grid, block>>>(f.size(), N, L, f.get_Array());
	hipDeviceSynchronize();
}

cuFFT::cuFFT(const int _dim, const int *_n, const int _BATCH) : dim(_dim), BATCH(_BATCH)
{
	n = new int[dim];
	for (size_t i = 0; i < dim; i++)
		n[i] = _n[i];

	int NX, NY, NZ;
	L = 1;

	switch (dim)
	{
	case 1:
		NX = n[0];
		N = NX;

		if (hipfftPlan1d(&planZ2Z, NX, HIPFFT_Z2Z, BATCH) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			return;
		}
		if (hipfftPlan1d(&planD2Z, NX, HIPFFT_D2Z, BATCH) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			return;
		}
		if (hipfftPlan1d(&planZ2D, NX, HIPFFT_Z2D, BATCH) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			return;
		}
		break;

	case 3:
		NX = n[0];
		NY = n[1];
		NZ = n[2];
		N = NX * NY * NZ;

		if (hipfftPlan3d(&planZ2Z, NX, NY, NZ, HIPFFT_Z2Z) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			return;
		}
		if (hipfftPlan3d(&planD2Z, NX, NY, NZ, HIPFFT_D2Z) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			return;
		}
		if (hipfftPlan3d(&planZ2D, NX, NY, NZ, HIPFFT_Z2D) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			return;
		}

		break;
	
	default:
		throw;
	}
}
cuFFT::~cuFFT()
{
	hipfftDestroy(planD2Z);
	hipfftDestroy(planZ2D);
	hipfftDestroy(planZ2Z);
	delete[] n;
}
void cuFFT::reset(const int _dim, const int *_n, double _L, const int _BATCH)
{
	dim = _dim;
	BATCH = _BATCH;
	L = _L;

	hipfftDestroy(planD2Z);
	hipfftDestroy(planZ2D);
	hipfftDestroy(planZ2Z);
	delete[] n;
	n = new int[dim];
	for (size_t i = 0; i < dim; i++)
		n[i] = _n[i];

	int NX, NY, NZ;

	switch (dim)
	{
	case 1:
		NX = n[0];
		N = NX;

		if (hipfftPlan1d(&planZ2Z, NX, HIPFFT_Z2Z, BATCH) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			return;
		}
		if (hipfftPlan1d(&planD2Z, NX, HIPFFT_D2Z, BATCH) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			return;
		}
		if (hipfftPlan1d(&planZ2D, NX, HIPFFT_Z2D, BATCH) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			return;
		}
		break;

	case 3:
		NX = n[0];
		NY = n[1];
		NZ = n[2];
		N = NX * NY * NZ;

		if (hipfftPlan3d(&planZ2Z, NX, NY, NZ, HIPFFT_Z2Z) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			return;
		}
		if (hipfftPlan3d(&planD2Z, NX, NY, NZ, HIPFFT_D2Z) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			return;
		}
		if (hipfftPlan3d(&planZ2D, NX, NY, NZ, HIPFFT_Z2D) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			return;
		}

		break;

	default:
		throw;
	}
}