#include "hip/hip_runtime.h"
﻿#include "stdafx.h"


__global__ void kernalStepSymplectic41_v2(const double dt, const double normT, cudaRVector3Dev k_sqr, cudaCVector3Dev Q, cudaCVector3Dev P, cudaCVector3Dev T)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < Q.size())
	{
		P(i) -= 0.67560359597982881702384390448573 * ((1 + k_sqr(i)) * Q(i) + T(i) * normT) * dt;
		Q(i) += 1.3512071919596576340476878089715 * P(i) * dt;
	}
}
__global__ void kernalStepSymplectic42_v2(const double dt, const double normT, cudaRVector3Dev k_sqr, cudaCVector3Dev Q, cudaCVector3Dev P, cudaCVector3Dev T)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < Q.size())
	{
		P(i) -= -0.17560359597982881702384390448573 * ((1 + k_sqr(i)) * Q(i) + T(i) * normT) * dt;
		Q(i) += -1.702414383919315268095375617943 * P(i) * dt;
	}
}
__global__ void kernalStepSymplectic43_v2(const double dt, const double normT, cudaRVector3Dev k_sqr, cudaCVector3Dev Q, cudaCVector3Dev P, cudaCVector3Dev T)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < Q.size())
	{
		P(i) -= -0.17560359597982881702384390448573 * ((1 + k_sqr(i)) * Q(i) + T(i) * normT) * dt;
		Q(i) += 1.3512071919596576340476878089715 * P(i) * dt;
	}
}
__global__ void kernalStepSymplectic44_v2(const double dt, const double normT, cudaRVector3Dev k_sqr, cudaCVector3Dev Q, cudaCVector3Dev P, cudaCVector3Dev T)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < Q.size())
	{
		P(i) -= 0.67560359597982881702384390448573 * ((1 + k_sqr(i)) * Q(i) + T(i) * normT) * dt;
	}
}

__global__ void kernel_Phi4_Phi6_v2(const int N, const double L, const double lambda, const double g, cudaRVector3Dev q, cudaRVector3Dev t)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	double f = q(i) / L;
	if (i < N)
	{
		t(i) = f * f * f * (lambda + g * f * f);
	}
}

equationsAxionSymplectic_3D::equationsAxionSymplectic_3D(hipStream_t _stream)
{
	stream = _stream;
}

void equationsAxionSymplectic_3D::equationCuda(const double dt, cudaGrid_3D& Grid)
{
	int N1 = (int)Grid.getN1();
	int N2 = (int)Grid.getN2();
	int N3red = (int)Grid.getN3red();
	int Nred = N1 * N2 * N3red;

	dim3 block(BLOCK_SIZE);
	dim3 grid((Nred + BLOCK_SIZE + 1) / BLOCK_SIZE);

	double normT = Grid.getVolume() / Grid.size();
	
	getNonlin_Phi4_Phi6(Grid);
	kernalStepSymplectic41_v2<<<grid, block, 0, stream>>>(dt, normT, Grid.get_k_sqr(), Grid.get_Q(), Grid.get_P(), Grid.get_T());
		
	getNonlin_Phi4_Phi6(Grid);
	kernalStepSymplectic42_v2<<<grid, block, 0, stream>>>(dt, normT, Grid.get_k_sqr(), Grid.get_Q(), Grid.get_P(), Grid.get_T());
	
	getNonlin_Phi4_Phi6(Grid);
	kernalStepSymplectic43_v2<<<grid, block, 0, stream>>>(dt, normT, Grid.get_k_sqr(), Grid.get_Q(), Grid.get_P(), Grid.get_T());
	
	getNonlin_Phi4_Phi6(Grid);
	kernalStepSymplectic44_v2<<<grid, block, 0, stream>>>(dt, normT, Grid.get_k_sqr(), Grid.get_Q(), Grid.get_P(), Grid.get_T());
	
	hipStreamSynchronize(stream);
	Grid.setSmthChanged();
	
	Grid.timestep(dt);
}

void equationsAxionSymplectic_3D::getNonlin_Phi4_Phi6(cudaGrid_3D& Grid)
{
	int N1 = (int)Grid.getN1();
	int N2 = (int)Grid.getN2();
	int N3 = (int)Grid.getN3();
	int N3red = (int)Grid.getN3red();
	int N = N1 * N2 * N3;

	dim3 block(BLOCK_SIZE);
	dim3 grid((N + BLOCK_SIZE + 1) / BLOCK_SIZE);

	bool isNormed = false;
	Grid.ifftQ(isNormed, true);
	kernel_Phi4_Phi6_v2<<<grid, block, 0, stream>>>(N, Grid.getVolume(), Grid.get_lambda(), Grid.get_g(), Grid.get_q(), Grid.get_t());
	Grid.doFFTforward(Grid.get_t(), Grid.get_T(), false);
}
