#include "hip/hip_runtime.h"
#include <>

#include "cudaVector.h"
#include "cudaFFT.h"

#include <hip/hip_runtime_api.h>

// #define CALBACKS
#define FFT_BLOCK_SIZE 128

#ifdef CALBACKS

__device__ void callbackForwardNormZ(void* dataOut, size_t offset, hipfftDoubleComplex element, void* callerInfo, void* sharedPointer)
{
	double* dataLN = (double*)callerInfo;
	complex el = *(complex*)&element;
	((complex*)dataOut)[offset] = el * (dataLN[0] / dataLN[1]);
}
__device__ void callbackForwardNormD(void* dataOut, size_t offset, hipfftDoubleReal element, void* callerInfo, void* sharedPointer)
{
	double* dataLN = (double*)callerInfo;
	((double*)dataOut)[offset] = element * (dataLN[0] / dataLN[1]);
}
__device__ void callbackInverseNormZ(void* dataOut, size_t offset, hipfftDoubleComplex element, void* callerInfo, void* sharedPointer)
{
	double* dataLN = (double*)callerInfo;
	complex el = *(complex*)&element;
	((complex*)dataOut)[offset] = el / dataLN[0];
}
__device__ void callbackInverseNormD(void* dataOut, size_t offset, hipfftDoubleReal element, void* callerInfo, void* sharedPointer)
{
	double* dataLN = (double*)callerInfo;
	((double*)dataOut)[offset] = element / dataLN[0];
}

__device__ hipfftCallbackStoreZ d_callbackForwardNormZ = callbackForwardNormZ;
__device__ hipfftCallbackStoreD d_callbackForwardNormD = callbackForwardNormD;
__device__ hipfftCallbackStoreZ d_callbackInverseNormZ = callbackInverseNormZ;
__device__ hipfftCallbackStoreD d_callbackInverseNormD = callbackInverseNormD;


__device__ void callbackForwardNormC(void* dataOut, size_t offset, hipfftComplex element, void* callerInfo, void* sharedPointer)
{
	float* dataVandN = (float*)callerInfo;
	complex el = *(complex*)&element;
	((complex*)dataOut)[offset] = el * (dataVandN[0] / dataVandN[1]);
}
__device__ void callbackForwardNormR(void* dataOut, size_t offset, hipfftReal element, void* callerInfo, void* sharedPointer)
{
	float* dataVandN = (float*)callerInfo;
	((float*)dataOut)[offset] = element * (dataVandN[0] / dataVandN[1]);
}
__device__ void callbackInverseNormC(void* dataOut, size_t offset, hipfftComplex element, void* callerInfo, void* sharedPointer)
{
	float* dataVandN = (float*)callerInfo;
	complex el = *(complex*)&element;
	((complex*)dataOut)[offset] = el / dataVandN[0];
}
__device__ void callbackInverseNormR(void* dataOut, size_t offset, hipfftReal element, void* callerInfo, void* sharedPointer)
{
	float* dataVandN = (float*)callerInfo;
	((float*)dataOut)[offset] = element / dataVandN[0];
}

__device__ hipfftCallbackStoreC d_callbackForwardNormC = callbackForwardNormC;
__device__ hipfftCallbackStoreR d_callbackForwardNormR = callbackForwardNormR;
__device__ hipfftCallbackStoreC d_callbackInverseNormC = callbackInverseNormC;
__device__ hipfftCallbackStoreR d_callbackInverseNormR = callbackInverseNormR;

#else

__global__ void kernelForwardNorm(const size_t size, const size_t N, const real vol, real* V)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		V[i] *= (vol / N);
	}
}
__global__ void kernelForwardNorm(const size_t size, const size_t N, const real vol, complex* V)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		V[i] *= (vol / N);
	}
}
__global__ void kernelInverseNorm(const size_t size, const size_t N, const real vol, real* V)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		V[i] /= vol;
	}
}
__global__ void kernelInverseNorm(const size_t size, const size_t N, const real vol, complex* V)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		V[i] /= vol;
	}
}

#endif // CALBACKS

void cuFFT::forward(cudaCVector3& f, cudaCVector3& F)
{
	checkCudaErrors(hipfftXtExec(planC2CF, f.getArray(), F.getArray(), HIPFFT_FORWARD));
	
#ifndef CALBACKS
	dim3 block(FFT_BLOCK_SIZE);
	dim3 grid((static_cast<unsigned int>(F.getSize()) + FFT_BLOCK_SIZE - 1) / FFT_BLOCK_SIZE);
	kernelForwardNorm <<< grid, block, 0, stream >>> (F.getSize(), N, volume, F.getArray());
#endif // !CALBACKS
}
void cuFFT::forward(cudaRVector3& f, cudaCVector3& F)
{
	checkCudaErrors(hipfftXtExec(planR2C, f.getArray(), F.getArray(), HIPFFT_FORWARD));

#ifndef CALBACKS
	dim3 block(FFT_BLOCK_SIZE);
	dim3 grid((static_cast<unsigned int>(F.getSize()) + FFT_BLOCK_SIZE - 1) / FFT_BLOCK_SIZE);
	kernelForwardNorm << < grid, block, 0, stream >> > (F.getSize(), N, volume, F.getArray());
#endif // !CALBACKS
}
void cuFFT::inverce(cudaCVector3& F, cudaCVector3& f)
{
	checkCudaErrors(hipfftXtExec(planC2CI, F.getArray(), f.getArray(), HIPFFT_BACKWARD));
	
#ifndef CALBACKS
	dim3 block(FFT_BLOCK_SIZE);
	dim3 grid((static_cast<unsigned int>(f.getSize()) + FFT_BLOCK_SIZE - 1) / FFT_BLOCK_SIZE);
	kernelInverseNorm << < grid, block, 0, stream >> > (f.getSize(), N, volume, f.getArray());
#endif // !CALBACKS
}
void cuFFT::inverce(cudaCVector3& F, cudaRVector3& f)
{
	checkCudaErrors(hipfftXtExec(planC2R, F.getArray(), f.getArray(), HIPFFT_BACKWARD));

#ifndef CALBACKS
	dim3 block(FFT_BLOCK_SIZE);
	dim3 grid((static_cast<unsigned int>(f.getSize()) + FFT_BLOCK_SIZE - 1) / FFT_BLOCK_SIZE);
	kernelInverseNorm <<< grid, block, 0, stream >>> (f.getSize(), N, volume, f.getArray());
#endif // !CALBACKS
}

void cuFFT::setStream(hipStream_t stream) 
{
	checkCudaErrors(hipfftSetStream(planC2CF, stream));
	checkCudaErrors(hipfftSetStream(planC2CI, stream));
	checkCudaErrors(hipfftSetStream(planR2C, stream));
	checkCudaErrors(hipfftSetStream(planC2R, stream));
}

cuFFT::cuFFT()
{
	isInitialized = false;
#ifdef CALBACKS
	checkCudaErrors(hipMemcpyFromSymbol(&h_callbackForwardNormC, HIP_SYMBOL(d_callbackForwardNormC), sizeof(h_callbackForwardNormC)));
	checkCudaErrors(hipMemcpyFromSymbol(&h_callbackForwardNormR, HIP_SYMBOL(d_callbackForwardNormR), sizeof(h_callbackForwardNormR)));
	checkCudaErrors(hipMemcpyFromSymbol(&h_callbackInverseNormC, HIP_SYMBOL(d_callbackInverseNormC), sizeof(h_callbackInverseNormC)));
	checkCudaErrors(hipMemcpyFromSymbol(&h_callbackInverseNormR, HIP_SYMBOL(d_callbackInverseNormR), sizeof(h_callbackInverseNormR)));

	checkCudaErrors(hipMemcpyFromSymbol(&h_callbackForwardNormZ, HIP_SYMBOL(d_callbackForwardNormZ), sizeof(h_callbackForwardNormZ)));
	checkCudaErrors(hipMemcpyFromSymbol(&h_callbackForwardNormD, HIP_SYMBOL(d_callbackForwardNormD), sizeof(h_callbackForwardNormD)));
	checkCudaErrors(hipMemcpyFromSymbol(&h_callbackInverseNormZ, HIP_SYMBOL(d_callbackInverseNormZ), sizeof(h_callbackInverseNormZ)));
	checkCudaErrors(hipMemcpyFromSymbol(&h_callbackInverseNormD, HIP_SYMBOL(d_callbackInverseNormD), sizeof(h_callbackInverseNormD)));
#endif // CALBACKS
}
cuFFT::cuFFT(const int _dim, const int *_n, real _volume, const int _BATCH, hipStream_t _stream) : dim(_dim), volume(_volume), BATCH(_BATCH), stream(_stream)
{
#ifdef CALBACKS
	checkCudaErrors(hipMemcpyFromSymbol(&h_callbackForwardNormC, HIP_SYMBOL(d_callbackForwardNormC), sizeof(h_callbackForwardNormC)));
	checkCudaErrors(hipMemcpyFromSymbol(&h_callbackForwardNormR, HIP_SYMBOL(d_callbackForwardNormR), sizeof(h_callbackForwardNormR)));
	checkCudaErrors(hipMemcpyFromSymbol(&h_callbackInverseNormC, HIP_SYMBOL(d_callbackInverseNormC), sizeof(h_callbackInverseNormC)));
	checkCudaErrors(hipMemcpyFromSymbol(&h_callbackInverseNormR, HIP_SYMBOL(d_callbackInverseNormR), sizeof(h_callbackInverseNormR)));

	checkCudaErrors(hipMemcpyFromSymbol(&h_callbackForwardNormZ, HIP_SYMBOL(d_callbackForwardNormZ), sizeof(h_callbackForwardNormZ)));
	checkCudaErrors(hipMemcpyFromSymbol(&h_callbackForwardNormD, HIP_SYMBOL(d_callbackForwardNormD), sizeof(h_callbackForwardNormD)));
	checkCudaErrors(hipMemcpyFromSymbol(&h_callbackInverseNormZ, HIP_SYMBOL(d_callbackInverseNormZ), sizeof(h_callbackInverseNormZ)));
	checkCudaErrors(hipMemcpyFromSymbol(&h_callbackInverseNormD, HIP_SYMBOL(d_callbackInverseNormD), sizeof(h_callbackInverseNormD)));
#endif // CALBACKS
	reset(_dim, _n, _volume, _BATCH, _stream);
}
cuFFT::~cuFFT()
{
	clear();
}
void cuFFT::reset(const int _dim, const int *_n, real _volume, const int _BATCH, hipStream_t _stream)
{
	clear();

	dim = _dim;
	n = new int[dim];
	N = 1;
	for (size_t i = 0; i < dim; i++) {
		n[i] = _n[i];
		N *= n[i];
	}
	
	BATCH = _BATCH;
	volume = _volume;

	switch (dim)
	{
	case 1:
		/*NX = n[0];
		N = NX;

		if (hipfftPlan1d(&planZ2Z, NX, HIPFFT_Z2Z, BATCH) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			throw;
		}
		if (hipfftPlan1d(&planR2C, NX, HIPFFT_D2Z, BATCH) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			throw;
		}
		if (hipfftPlan1d(&planC2R, NX, HIPFFT_Z2D, BATCH) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			throw;
		}
		break;*/
		throw -1;
	case 3:		
#ifdef CALBACKS
		std::cout << "Callbacks enabled" << std::endl;

		size_t workSize;

		checkCudaErrors(hipfftCreate(&planC2CF));
		checkCudaErrors(hipfftCreate(&planC2CI));
		checkCudaErrors(hipfftCreate(&planR2C));
		checkCudaErrors(hipfftCreate(&planC2R));

		checkCudaErrors(hipMallocManaged(&callbackData, 2 * sizeof(real)));
		callbackData[0] = volume;
		callbackData[1] = (real)N;

		if (typeid(real) == typeid(double)) {
			checkCudaErrors(hipfftMakePlan3d(planC2CF, n[0], n[1], n[2], HIPFFT_Z2Z, &workSize));
			checkCudaErrors(hipfftMakePlan3d(planC2CI, n[0], n[1], n[2], HIPFFT_Z2Z, &workSize));
			checkCudaErrors(hipfftMakePlan3d(planR2C, n[0], n[1], n[2], HIPFFT_D2Z, &workSize));
			checkCudaErrors(hipfftMakePlan3d(planC2R, n[0], n[1], n[2], HIPFFT_Z2D, &workSize));
			checkCudaErrors(hipfftXtSetCallback(planC2CF, (void**)&h_callbackForwardNormZ, HIPFFT_CB_ST_COMPLEX_DOUBLE, (void**)&callbackData));
			checkCudaErrors(hipfftXtSetCallback(planC2CI, (void**)&h_callbackInverseNormZ, HIPFFT_CB_ST_COMPLEX_DOUBLE, (void**)&callbackData));
			checkCudaErrors(hipfftXtSetCallback(planR2C, (void**)&h_callbackForwardNormZ, HIPFFT_CB_ST_COMPLEX_DOUBLE, (void**)&callbackData));
			checkCudaErrors(hipfftXtSetCallback(planC2R, (void**)&h_callbackInverseNormD, HIPFFT_CB_ST_REAL_DOUBLE, (void**)&callbackData));
		}
		else {
			if (typeid(real) == typeid(float)) {
				checkCudaErrors(hipfftMakePlan3d(planC2CF, n[0], n[1], n[2], HIPFFT_C2C, &workSize));
				checkCudaErrors(hipfftMakePlan3d(planC2CI, n[0], n[1], n[2], HIPFFT_C2C, &workSize));
				checkCudaErrors(hipfftMakePlan3d(planR2C, n[0], n[1], n[2], HIPFFT_R2C, &workSize));
				checkCudaErrors(hipfftMakePlan3d(planC2R, n[0], n[1], n[2], HIPFFT_C2R, &workSize));
				checkCudaErrors(hipfftXtSetCallback(planC2CF, (void**)&h_callbackForwardNormC, HIPFFT_CB_ST_COMPLEX, (void**)&callbackData));
				checkCudaErrors(hipfftXtSetCallback(planC2CI, (void**)&h_callbackInverseNormC, HIPFFT_CB_ST_COMPLEX, (void**)&callbackData));
				checkCudaErrors(hipfftXtSetCallback(planR2C, (void**)&h_callbackForwardNormC, HIPFFT_CB_ST_COMPLEX, (void**)&callbackData));
				checkCudaErrors(hipfftXtSetCallback(planC2R, (void**)&h_callbackInverseNormR, HIPFFT_CB_ST_REAL, (void**)&callbackData));
			}
			else {
				throw - 1;
			}
		}
#else
		std::cout << "Callbacks disabled" << std::endl;

		if (typeid(real) == typeid(double)) {
			checkCudaErrors(hipfftPlan3d(&planC2CF, n[0], n[1], n[2], HIPFFT_Z2Z));
			checkCudaErrors(hipfftPlan3d(&planC2CI, n[0], n[1], n[2], HIPFFT_Z2Z));
			checkCudaErrors(hipfftPlan3d(&planR2C, n[0], n[1], n[2], HIPFFT_D2Z));
			checkCudaErrors(hipfftPlan3d(&planC2R, n[0], n[1], n[2], HIPFFT_Z2D));
		}
		else {
			if (typeid(real) == typeid(float)) {
				checkCudaErrors(hipfftPlan3d(&planC2CF, n[0], n[1], n[2], HIPFFT_C2C));
				checkCudaErrors(hipfftPlan3d(&planC2CI, n[0], n[1], n[2], HIPFFT_C2C));
				checkCudaErrors(hipfftPlan3d(&planR2C, n[0], n[1], n[2], HIPFFT_R2C));
				checkCudaErrors(hipfftPlan3d(&planC2R, n[0], n[1], n[2], HIPFFT_C2R));
			}
			else {
				throw - 1;
			}
		}
#endif // CALBACKS
		break;
	default:
		throw -1;
	}

	stream = _stream;
	setStream(stream);

	isInitialized = true;
}

void cuFFT::clear()
{
	if (isInitialized) {
		delete[] n;
		checkCudaErrors(hipfftDestroy(planC2CF));
		checkCudaErrors(hipfftDestroy(planC2CI));
		checkCudaErrors(hipfftDestroy(planR2C));
		checkCudaErrors(hipfftDestroy(planC2R));
#ifdef CALBACKS
		checkCudaErrors(hipFree(callbackData));
#endif // CALBACKS
		isInitialized = false;
	}
}