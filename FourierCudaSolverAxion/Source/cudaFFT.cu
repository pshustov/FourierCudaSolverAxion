#include "hip/hip_runtime.h"
#include <>

#include "cudaVector.h"
#include "cudaFFT.h"

#include <hip/hip_runtime_api.h>

#define FFT_BLOCK_SIZE 128

#ifdef __linux__

__device__ void callbackForwardNormZ(void* dataOut, size_t offset, hipfftDoubleComplex element, void* callerInfo, void* sharedPointer)
{
	double* dataLN = (double*)callerInfo;
	complex el = *(complex*)&element;
	((complex*)dataOut)[offset] = el * (dataLN[0] / dataLN[1]);
}
__device__ void callbackForwardNormD(void* dataOut, size_t offset, hipfftDoubleReal element, void* callerInfo, void* sharedPointer)
{
	double* dataLN = (double*)callerInfo;
	((double*)dataOut)[offset] = element * (dataLN[0] / dataLN[1]);
}
__device__ void callbackInverseNormZ(void* dataOut, size_t offset, hipfftDoubleComplex element, void* callerInfo, void* sharedPointer)
{
	double* dataLN = (double*)callerInfo;
	complex el = *(complex*)&element;
	((complex*)dataOut)[offset] = el / dataLN[0];
}
__device__ void callbackInverseNormD(void* dataOut, size_t offset, hipfftDoubleReal element, void* callerInfo, void* sharedPointer)
{
	double* dataLN = (double*)callerInfo;
	((double*)dataOut)[offset] = element / dataLN[0];
}

__device__ hipfftCallbackStoreZ d_callbackForwardNormZ = callbackForwardNormZ;
__device__ hipfftCallbackStoreD d_callbackForwardNormD = callbackForwardNormD;
__device__ hipfftCallbackStoreZ d_callbackInverseNormZ = callbackInverseNormZ;
__device__ hipfftCallbackStoreD d_callbackInverseNormD = callbackInverseNormD;

#else

__global__ void kernelForwardNorm(const size_t size, const size_t N, const real L, real* V)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		V[i] = V[i] * L / N;
	}
}
__global__ void kernelForwardNorm(const size_t size, const size_t N, const real L, complex* V)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		V[i] = V[i] * L / N;
	}
}
__global__ void kernelInverseNorm(const size_t size, const size_t N, const real L, real* V)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		V[i] = V[i] / L;
	}
}
__global__ void kernelInverseNorm(const size_t size, const size_t N, const real L, complex* V)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		V[i] = V[i] / L;
	}
}

#endif // __linux__

void cuFFT::forward(cudaCVector3& f, cudaCVector3& F)
{
	checkCudaErrors(hipfftXtExec(planC2CF, f.getArray(), F.getArray(), HIPFFT_FORWARD));
	
#ifndef __linux__
	dim3 block(FFT_BLOCK_SIZE);
	dim3 grid((static_cast<unsigned int>(F.size()) + FFT_BLOCK_SIZE - 1) / FFT_BLOCK_SIZE);
	kernelForwardNorm <<< grid, block, 0, stream >>> (F.size(), N, L, F.getArray());
#endif // !__linux__
}
void cuFFT::forward(cudaRVector3& f, cudaCVector3& F)
{
	checkCudaErrors(hipfftXtExec(planR2C, f.getArray(), F.getArray(), HIPFFT_FORWARD));

#ifndef __linux__
	dim3 block(FFT_BLOCK_SIZE);
	dim3 grid((static_cast<unsigned int>(F.size()) + FFT_BLOCK_SIZE - 1) / FFT_BLOCK_SIZE);
	kernelForwardNorm << < grid, block, 0, stream >> > (F.size(), N, L, F.getArray());
#endif // !__linux__
}
void cuFFT::inverce(cudaCVector3& F, cudaCVector3& f)
{
	checkCudaErrors(hipfftXtExec(planC2CI, F.getArray(), f.getArray(), HIPFFT_BACKWARD));
	
#ifndef __linux__
	dim3 block(FFT_BLOCK_SIZE);
	dim3 grid((static_cast<unsigned int>(f.size()) + FFT_BLOCK_SIZE - 1) / FFT_BLOCK_SIZE);
	kernelInverseNorm << < grid, block, 0, stream >> > (f.size(), N, L, f.getArray());
#endif // !__linux__
}
void cuFFT::inverce(cudaCVector3& F, cudaRVector3& f)
{
	checkCudaErrors(hipfftXtExec(planC2R, F.getArray(), f.getArray(), HIPFFT_BACKWARD));
	//checkCudaErrors(hipfftExecC2R(planC2R, (hipfftComplex*)F.getArray(), (hipfftReal*)f.getArray()));
	//checkCudaErrors(hipfftExecC2R(planC2R, (hipfftComplex*)F.getArray(), (hipfftReal*)f.getArray()));


#ifndef __linux__
	dim3 block(FFT_BLOCK_SIZE);
	dim3 grid((static_cast<unsigned int>(f.size()) + FFT_BLOCK_SIZE - 1) / FFT_BLOCK_SIZE);
	kernelInverseNorm << < grid, block, 0, stream >> > (f.size(), N, L, f.getArray());
#endif // !__linux__
}

void cuFFT::setStream(hipStream_t stream) 
{
	checkCudaErrors(hipfftSetStream(planC2CF, stream));
	checkCudaErrors(hipfftSetStream(planC2CI, stream));
	checkCudaErrors(hipfftSetStream(planR2C, stream));
	checkCudaErrors(hipfftSetStream(planC2R, stream));
}

cuFFT::cuFFT(hipStream_t _stream) : stream(_stream)
{
	dim = 1;
	n = new int[dim];
	n[0] = 1024;
	L = 10;
	N = 1024;

	BATCH = 1;

	checkCudaErrors(hipfftCreate(&planC2CF));
	checkCudaErrors(hipfftCreate(&planC2CI));
	checkCudaErrors(hipfftCreate(&planR2C));
	checkCudaErrors(hipfftCreate(&planC2R));

	setStream(stream);

#ifdef __linux__
	std::cout << "LINUX detected" << std::endl;
	checkCudaErrors(hipMemcpyFromSymbol(&h_callbackForwardNormZ, HIP_SYMBOL(d_callbackForwardNormZ), sizeof(h_callbackForwardNormZ)));
	checkCudaErrors(hipMemcpyFromSymbol(&h_callbackForwardNormD, HIP_SYMBOL(d_callbackForwardNormD), sizeof(h_callbackForwardNormD)));
	checkCudaErrors(hipMemcpyFromSymbol(&h_callbackInverseNormZ, HIP_SYMBOL(d_callbackInverseNormZ), sizeof(h_callbackInverseNormZ)));
	checkCudaErrors(hipMemcpyFromSymbol(&h_callbackInverseNormD, HIP_SYMBOL(d_callbackInverseNormD), sizeof(h_callbackInverseNormD)));
#endif

	checkCudaErrors(hipMallocManaged(&callbackData, 2 * sizeof(double)));
	callbackData[0] = L;
	callbackData[1] = N;
}

cuFFT::cuFFT(const int _dim, const int *_n, real _L, const int _BATCH, hipStream_t _stream) : dim(_dim), BATCH(_BATCH), stream(_stream)
{
	dim = 1;
	n = new int[dim];
	n[0] = 1024;
	L = 10;
	N = 1024;

	BATCH = 1;

	checkCudaErrors(hipfftCreate(&planC2CF));
	checkCudaErrors(hipfftCreate(&planC2CI));
	checkCudaErrors(hipfftCreate(&planR2C));
	checkCudaErrors(hipfftCreate(&planC2R));

	setStream(stream);

#ifdef __linux__
	std::cout << "LINUX detected" << std::endl;
	checkCudaErrors(hipMemcpyFromSymbol(&h_callbackForwardNormZ, HIP_SYMBOL(d_callbackForwardNormZ), sizeof(h_callbackForwardNormZ)));
	checkCudaErrors(hipMemcpyFromSymbol(&h_callbackForwardNormD, HIP_SYMBOL(d_callbackForwardNormD), sizeof(h_callbackForwardNormD)));
	checkCudaErrors(hipMemcpyFromSymbol(&h_callbackInverseNormZ, HIP_SYMBOL(d_callbackInverseNormZ), sizeof(h_callbackInverseNormZ)));
	checkCudaErrors(hipMemcpyFromSymbol(&h_callbackInverseNormD, HIP_SYMBOL(d_callbackInverseNormD), sizeof(h_callbackInverseNormD)));
#endif

	checkCudaErrors(hipMallocManaged(&callbackData, 2 * sizeof(double)));
	callbackData[0] = L;
	callbackData[1] = N;

	reset(_dim, _n, _L, _BATCH, _stream);
}
cuFFT::~cuFFT()
{
	checkCudaErrors(hipfftDestroy(planC2CF));
	checkCudaErrors(hipfftDestroy(planC2CI));
	checkCudaErrors(hipfftDestroy(planR2C));
	checkCudaErrors(hipfftDestroy(planC2R));
	checkCudaErrors(hipFree(callbackData));
	delete[] n;
}
void cuFFT::reset(const int _dim, const int *_n, real _L, const int _BATCH, hipStream_t _stream)
{
	dim = _dim;
	delete[] n;
	n = new int[dim];
	N = 1;
	for (size_t i = 0; i < dim; i++) {
		n[i] = _n[i];
		N *= n[i];
	}
	
	BATCH = _BATCH;
	L = _L;

	callbackData[0] = L;
	callbackData[1] = N;

	checkCudaErrors(hipfftDestroy(planC2CF));
	checkCudaErrors(hipfftDestroy(planC2CI));
	checkCudaErrors(hipfftDestroy(planR2C));
	checkCudaErrors(hipfftDestroy(planC2R));

	switch (dim)
	{
	case 1:
		/*NX = n[0];
		N = NX;

		if (hipfftPlan1d(&planZ2Z, NX, HIPFFT_Z2Z, BATCH) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			throw;
		}
		if (hipfftPlan1d(&planR2C, NX, HIPFFT_D2Z, BATCH) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			throw;
		}
		if (hipfftPlan1d(&planC2R, NX, HIPFFT_Z2D, BATCH) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			throw;
		}
		break;*/
		throw;

	case 3:
		size_t workSize;
		
#ifdef _WIN64
		if (typeid(real)==typeid(double)) {
			checkCudaErrors(hipfftPlan3d(&planC2CF, n[0], n[1], n[2], HIPFFT_Z2Z));
			checkCudaErrors(hipfftPlan3d(&planC2CI, n[0], n[1], n[2], HIPFFT_Z2Z));
			checkCudaErrors(hipfftPlan3d(&planR2C, n[0], n[1], n[2], HIPFFT_D2Z));
			checkCudaErrors(hipfftPlan3d(&planC2R, n[0], n[1], n[2], HIPFFT_Z2D));
		}
		else {
			if (typeid(real) == typeid(float)) {
				checkCudaErrors(hipfftPlan3d(&planC2CF, n[0], n[1], n[2], HIPFFT_C2C));
				checkCudaErrors(hipfftPlan3d(&planC2CI, n[0], n[1], n[2], HIPFFT_C2C));
				checkCudaErrors(hipfftPlan3d(&planR2C, n[0], n[1], n[2], HIPFFT_R2C));
				checkCudaErrors(hipfftPlan3d(&planC2R, n[0], n[1], n[2], HIPFFT_C2R));
			}
			else {
				throw;
			}
		}

#endif // _WIN64


#ifdef __linux__
		checkCudaErrors(hipfftCreate(&planC2CF));
		checkCudaErrors(hipfftMakePlan3d(planC2CF, n[0], n[1], n[2], HIPFFT_Z2Z, &workSize));

		checkCudaErrors(hipfftCreate(&planC2CI));
		checkCudaErrors(hipfftMakePlan3d(planC2CI, n[0], n[1], n[2], HIPFFT_Z2Z, &workSize));

		checkCudaErrors(hipfftCreate(&planR2C));
		checkCudaErrors(hipfftMakePlan3d(planR2C, n[0], n[1], n[2], HIPFFT_D2Z, &workSize));

		checkCudaErrors(hipfftCreate(&planC2R));
		checkCudaErrors(hipfftMakePlan3d(planC2R, n[0], n[1], n[2], HIPFFT_Z2D, &workSize));

		std::cout << "LINUX detected" << std::endl;
		checkCudaErrors(hipfftXtSetCallback(planC2CF, (void**)&h_callbackForwardNormZ, HIPFFT_CB_ST_COMPLEX_DOUBLE, (void**)&callbackData));
		checkCudaErrors(hipfftXtSetCallback(planC2CI, (void**)&h_callbackInverseNormZ, HIPFFT_CB_ST_COMPLEX_DOUBLE, (void**)&callbackData));
		checkCudaErrors(hipfftXtSetCallback(planR2C, (void**)&h_callbackForwardNormZ, HIPFFT_CB_ST_COMPLEX_DOUBLE, (void**)&callbackData));
		checkCudaErrors(hipfftXtSetCallback(planC2R, (void**)&h_callbackInverseNormD, HIPFFT_CB_ST_REAL_DOUBLE, (void**)&callbackData));
#endif	// __linux__

		break;

	default:
		throw;
	}

	stream = _stream;
	setStream(stream);
}