#include "hip/hip_runtime.h"
#include "stdafx.h"

__device__ double getOmega(double lam, double g, double f2mean, double k_sqr)
{
	return sqrt(1 + k_sqr + 3 * lam * f2mean + 15 * g * f2mean * f2mean);
}

__global__ void setQquad(cudaCVector3Dev Q) 
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	size_t j = blockIdx.y * blockDim.y + threadIdx.y;
	size_t k = blockIdx.z * blockDim.z + threadIdx.z;
	
	size_t N1 = Q.getN1(), N2 = Q.getN2(), N3 = Q.getN3();

	if (i < N1 && j < N2 && k < N3)
	{
		size_t ind = (i * N2 + j) * N3 + k;
		Q(ind) = Q(ind).absSqr();
		//if (k == 0) {
		//	Q(ind) *= Q(ind).get_conj();
		//}
		//else {
		//	Q(ind) *= 2 * Q(ind).get_conj();
		//}
	}

}

__global__ void kernelSetDistributionFunction(double lam, double g, double f2mean, cudaRVector3Dev kSqr, cudaCVector3Dev Q, cudaCVector3Dev P) 
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	size_t j = blockIdx.y * blockDim.y + threadIdx.y;
	size_t k = blockIdx.z * blockDim.z + threadIdx.z;

	size_t N1 = kSqr.getN1(), N2 = kSqr.getN2(), N3 = kSqr.getN3();
	
	if (i < N1 && j < N2 && k < N3)
	{
		size_t ind = (i * N2 + j) * N3 + k;
		double omega = getOmega(lam, g, f2mean, kSqr(ind));
		double m = 1 + kSqr(ind) + 1.5 * lam * f2mean + 5 * g * f2mean * f2mean;

		if (k == 0) {
			P(ind) = 0.5 * (P(ind).absSqr() + m * Q(ind)) / omega;
			//P(ind) = 0;
			Q(ind) = sqrt(kSqr(ind))* P(ind);
		}
		else {
			P(ind) = (P(ind).absSqr() + m * Q(ind)) / omega;
			//P(ind) = 0;
			Q(ind) = sqrt(kSqr(ind))* P(ind);
		}
	}
}


void Distribution::setDistribution(cudaGrid_3D& Grid)
{
	outFile.open("outNumberAndMomentum.txt");

	time = Grid.get_time();
	lam = Grid.get_lambda();
	g = Grid.get_g();
	volume = Grid.getVolume();
	f2mean = 0;
	k_sqr = Grid.get_k_sqr();
	Q = Grid.get_Q();
	P = Grid.get_P();

	hipStreamCreate(&streamDistrib);
}


void Distribution::calculateNumberAndMomentum()
{
	size_t Bx = 16, By = 8, Bz = 1;
	dim3 block3(Bx, By, Bz);
	dim3 grid3Red((Q.getN1() + Bx - 1) / Bx, (Q.getN2() + By - 1) / By, (Q.getN3() + Bz - 1) / Bz);

	dim3 block(BLOCK_SIZE);
	dim3 grid((k_sqr.size() + BLOCK_SIZE - 1) / BLOCK_SIZE);


	setQquad<<< grid3Red, block3, 0, streamDistrib >>>(Q);
	hipStreamSynchronize(streamDistrib);

	complex f2m = Q.getSum(streamDistrib).real() / (volume * volume);
	f2mean = f2m.real();

	if ((1 + 3 * lam * f2mean + 15 * g * f2mean * f2mean) < 0) {
		numberOfParticles = -1;
		meanMomentum = -1;

		if (!isAlarmed) {
			std::cout << "!!! Unstable condition !!!" << std::endl;
			isAlarmed = true;
		}
	}
	else
	{
		kernelSetDistributionFunction<<< grid3Red, block3, 0, streamDistrib >>>(lam, g, f2mean, k_sqr, Q, P);
		hipStreamSynchronize(streamDistrib);

		numberOfParticles = P.getSum(streamDistrib).real() / volume;
		meanMomentum = Q.getSum(streamDistrib).real() / (volume * numberOfParticles);

		isAlarmed = false;
	}

	outFile << time << '\t' << numberOfParticles << '\t' << meanMomentum << std::endl;
}

