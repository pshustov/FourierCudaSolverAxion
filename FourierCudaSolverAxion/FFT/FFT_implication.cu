#include "hip/hip_runtime.h"
#include "stdafx.h"

__global__ void kernelForwardNorm(const size_t size, const size_t N, const double L, double *V)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		V[i] = V[i] * L / N;
	}
}
__global__ void kernelForwardNorm(const size_t size, const size_t N, const double L, complex *V)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		V[i] = V[i] * L / N;
	}
}
__global__ void kernelInverseNorm(const size_t size, const size_t N, const double L, double* V)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		V[i] = V[i] / L;
	}
}
__global__ void kernelInverseNorm(const size_t size, const size_t N, const double L, complex* V)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		V[i] = V[i] / L;
	}
}

void cuFFT::forward(cudaCVector &f, cudaCVector &F)
{
	if (hipfftExecZ2Z(planZ2Z, (hipfftDoubleComplex*)f.getArray(), (hipfftDoubleComplex*)F.getArray(), HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: ExecZ2Z Forward failed");
		return;
	}
	hipDeviceSynchronize();
	
	dim3 block(BLOCK_SIZE);
	dim3 grid((unsigned int)ceil((double)F.getN() / (double)BLOCK_SIZE));
	kernelForwardNorm<<<grid, block>>>(F.getN(), N, L, F.getArray());
	hipDeviceSynchronize();
}
void cuFFT::forward(cudaRVector &f, cudaCVector &F)
{
	if (hipfftExecD2Z(planD2Z, (hipfftDoubleReal*)f.getArray(), (hipfftDoubleComplex*)F.getArray()) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: ExecD2Z Forward failed");
		return;
	}
	hipDeviceSynchronize();
	
	dim3 block(BLOCK_SIZE);
	dim3 grid((unsigned int)ceil((double)F.getN() / (double)BLOCK_SIZE));
	kernelForwardNorm<<<grid, block>>>(F.getN(), N, L, F.getArray());
	hipDeviceSynchronize();
}
void cuFFT::forward(cudaCVector3 &f, cudaCVector3 &F)
{
	if (hipfftExecZ2Z(planZ2Z, (hipfftDoubleComplex*)f.getArray(), (hipfftDoubleComplex*)F.getArray(), HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: 3D ExecZ2Z Forward failed");
		return;
	}
	hipDeviceSynchronize();
	
	dim3 block(BLOCK_SIZE);
	dim3 grid((unsigned int)ceil((double)F.size() / (double)BLOCK_SIZE));
	kernelForwardNorm<<<grid, block>>>(F.size(), N, L, F.getArray());
	hipDeviceSynchronize();
}
void cuFFT::forward(cudaRVector3 &f, cudaCVector3 &F)
{
	if (hipfftExecD2Z(planD2Z, (hipfftDoubleReal*)f.getArray(), (hipfftDoubleComplex*)F.getArray()) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: 3D ExecD2Z Forward failed");
		return;
	}
	hipDeviceSynchronize();
	
	dim3 block(BLOCK_SIZE);
	dim3 grid((unsigned int)ceil((double)F.size() / (double)BLOCK_SIZE));
	kernelForwardNorm<<<grid, block>>>(F.size(), N, L, F.getArray());
	hipDeviceSynchronize();
}

void cuFFT::inverce(cudaCVector &F, cudaCVector &f)
{
	if (hipfftExecZ2Z(planZ2Z, (hipfftDoubleComplex*)F.getArray(), (hipfftDoubleComplex*)f.getArray(), HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: ExecZ2Z Inverce failed");
		return;
	}
	hipDeviceSynchronize();

	dim3 block(BLOCK_SIZE);
	dim3 grid((unsigned int)ceil((double)f.getN() / (double)BLOCK_SIZE));
	kernelInverseNorm<<<grid, block>>>(f.getN(), N, L, f.getArray());
	hipDeviceSynchronize();
}
void cuFFT::inverce(cudaCVector &F, cudaRVector &f)
{
	if (hipfftExecZ2D(planZ2D, (hipfftDoubleComplex*)F.getArray(), (hipfftDoubleReal*)f.getArray()) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: ExecZ2Z Inverce failed");
		return;
	}
	hipDeviceSynchronize();

	dim3 block(BLOCK_SIZE);
	dim3 grid((unsigned int)ceil((double)f.getN() / (double)BLOCK_SIZE));
	kernelInverseNorm<<<grid, block>>>(f.getN(), N, L, f.getArray());
	hipDeviceSynchronize();
}
void cuFFT::inverce(cudaCVector3 &F, cudaCVector3 &f)
{
	if (hipfftExecZ2Z(planZ2Z, (hipfftDoubleComplex*)F.getArray(), (hipfftDoubleComplex*)f.getArray(), HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: 3D ExecZ2Z Inverce failed");
		return;
	}
	hipDeviceSynchronize();

	dim3 block(BLOCK_SIZE);
	dim3 grid((unsigned int)ceil((double)f.size() / (double)BLOCK_SIZE));
	kernelInverseNorm<<<grid, block>>>(f.size(), N, L, f.getArray());
	hipDeviceSynchronize();
}
void cuFFT::inverce(cudaCVector3 &F, cudaRVector3 &f)
{
	if (hipfftExecZ2D(planZ2D, (hipfftDoubleComplex*)F.getArray(), (hipfftDoubleReal*)f.getArray()) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: 3D ExecZ2Z Inverce failed");
		return;
	}
	hipDeviceSynchronize();

	dim3 block(BLOCK_SIZE);
	dim3 grid((unsigned int)ceil((double)f.size() / (double)BLOCK_SIZE));
	kernelInverseNorm<<<grid, block>>>(f.size(), N, L, f.getArray());
	hipDeviceSynchronize();
}

cuFFT::cuFFT(const int _dim, const int *_n, const int _BATCH) : dim(_dim), BATCH(_BATCH)
{
	n = new int[dim];
	for (size_t i = 0; i < dim; i++)
		n[i] = _n[i];

	int NX, NY, NZ;
	L = 1;

	switch (dim)
	{
	case 1:
		NX = n[0];
		N = NX;

		if (hipfftPlan1d(&planZ2Z, NX, HIPFFT_Z2Z, BATCH) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			return;
		}
		if (hipfftPlan1d(&planD2Z, NX, HIPFFT_D2Z, BATCH) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			return;
		}
		if (hipfftPlan1d(&planZ2D, NX, HIPFFT_Z2D, BATCH) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			return;
		}
		break;

	case 3:
		NX = n[0];
		NY = n[1];
		NZ = n[2];
		N = NX * NY * NZ;

		if (hipfftPlan3d(&planZ2Z, NX, NY, NZ, HIPFFT_Z2Z) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			return;
		}
		if (hipfftPlan3d(&planD2Z, NX, NY, NZ, HIPFFT_D2Z) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			return;
		}
		if (hipfftPlan3d(&planZ2D, NX, NY, NZ, HIPFFT_Z2D) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			return;
		}

		break;
	
	default:
		throw;
	}
}
cuFFT::~cuFFT()
{
	hipfftDestroy(planD2Z);
	hipfftDestroy(planZ2D);
	hipfftDestroy(planZ2Z);
	delete[] n;
}
void cuFFT::reset(const int _dim, const int *_n, double _L, const int _BATCH)
{
	dim = _dim;
	BATCH = _BATCH;
	L = _L;

	hipfftDestroy(planD2Z);
	hipfftDestroy(planZ2D);
	hipfftDestroy(planZ2Z);
	delete[] n;
	n = new int[dim];
	for (size_t i = 0; i < dim; i++)
		n[i] = _n[i];

	int NX, NY, NZ;

	switch (dim)
	{
	case 1:
		NX = n[0];
		N = NX;

		if (hipfftPlan1d(&planZ2Z, NX, HIPFFT_Z2Z, BATCH) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			return;
		}
		if (hipfftPlan1d(&planD2Z, NX, HIPFFT_D2Z, BATCH) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			return;
		}
		if (hipfftPlan1d(&planZ2D, NX, HIPFFT_Z2D, BATCH) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			return;
		}
		break;

	case 3:
		NX = n[0];
		NY = n[1];
		NZ = n[2];
		N = NX * NY * NZ;

		if (hipfftPlan3d(&planZ2Z, NX, NY, NZ, HIPFFT_Z2Z) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			return;
		}
		if (hipfftPlan3d(&planD2Z, NX, NY, NZ, HIPFFT_D2Z) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			return;
		}
		if (hipfftPlan3d(&planZ2D, NX, NY, NZ, HIPFFT_Z2D) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			return;
		}

		break;

	default:
		throw;
	}
}