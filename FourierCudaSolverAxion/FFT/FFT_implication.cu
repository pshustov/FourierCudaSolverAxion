#include "hip/hip_runtime.h"
#include "stdafx.h"

#ifdef _WIN64
__global__ void kernelForwardNorm(const size_t size, const size_t N, const double L, double *V)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		V[i] = V[i] * L / N;
	}
}
__global__ void kernelForwardNorm(const size_t size, const size_t N, const double L, complex *V)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		V[i] = V[i] * L / N;
	}
}
__global__ void kernelInverseNorm(const size_t size, const size_t N, const double L, double* V)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		V[i] = V[i] / L;
	}
}
__global__ void kernelInverseNorm(const size_t size, const size_t N, const double L, complex* V)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		V[i] = V[i] / L;
	}
}
#endif

#ifdef linux
__device__ void callbackForwardNormZ(void* dataOut, size_t offset, hipfftDoubleComplex element, void* callerInfo, void* sharedPointer)
{
	double* dataLN = (double*)callerInfo;
	((complex*)dataOut)[offset] *= (dataLN[0] / dataLN[1]);
}
__device__ void callbackForwardNormD(void* dataOut, size_t offset, hipfftDoubleReal element, void* callerInfo, void* sharedPointer)
{
	double* dataLN = (double*)callerInfo;
	((double*)dataOut)[offset] *= (dataLN[0] / dataLN[1]);
}
__device__ void callbackInverseNormZ(void* dataOut, size_t offset, hipfftDoubleComplex element, void* callerInfo, void* sharedPointer)
{
	double* dataLN = (double*)callerInfo;
	((complex*)dataOut)[offset] /= dataLN[0];
}
__device__ void callbackInverseNormD(void* dataOut, size_t offset, hipfftDoubleReal element, void* callerInfo, void* sharedPointer)
{
	double* dataLN = (double*)callerInfo;
	((double*)dataOut)[offset] /= dataLN[0];
}

__device__ hipfftCallbackStoreZ d_callbackForwardNormZ = callbackForwardNormZ;
__device__ hipfftCallbackStoreD d_callbackForwardNormD = callbackForwardNormD;
__device__ hipfftCallbackStoreZ d_callbackInverseNormZ = callbackInverseNormZ;
__device__ hipfftCallbackStoreD d_callbackInverseNormD = callbackInverseNormD;
#endif

void cuFFT::forward(cudaCVector3& f, cudaCVector3& F, bool isNormed)
{
	checkCudaErrors(hipfftExecZ2Z(planZ2ZF, (hipfftDoubleComplex*)f.getArray(), (hipfftDoubleComplex*)F.getArray(), HIPFFT_FORWARD));
	checkCudaErrors(hipStreamSynchronize(stream));

#ifdef _WIN64
#endif
}
void cuFFT::forward(cudaRVector3& f, cudaCVector3& F, bool isNormed)
{
	checkCudaErrors(hipfftExecD2Z(planD2Z, (hipfftDoubleReal*)f.getArray(), (hipfftDoubleComplex*)F.getArray()));
	checkCudaErrors(hipStreamSynchronize(stream));
}
void cuFFT::inverce(cudaCVector3 &F, cudaCVector3 &f, bool isNormed)
{
	checkCudaErrors(hipfftExecZ2Z(planZ2ZI, (hipfftDoubleComplex*)F.getArray(), (hipfftDoubleComplex*)f.getArray(), HIPFFT_BACKWARD));
	checkCudaErrors(hipStreamSynchronize(stream));
}
void cuFFT::inverce(cudaCVector3 &F, cudaRVector3 &f, bool isNormed)
{
	checkCudaErrors(hipfftExecZ2D(planZ2D, (hipfftDoubleComplex*)F.getArray(), (hipfftDoubleReal*)f.getArray()));
	checkCudaErrors(hipStreamSynchronize(stream));
}


cuFFT::cuFFT(hipStream_t _stream) : stream(_stream)
{
	dim = 1;
	n = new int[dim];
	n[0] = 1024;
	L = 10;
	N = 1024;

	BATCH = 1;

	checkCudaErrors(hipfftCreate(&planZ2ZF));
	checkCudaErrors(hipfftCreate(&planZ2ZI));
	checkCudaErrors(hipfftCreate(&planD2Z));
	checkCudaErrors(hipfftCreate(&planZ2D));

	setStream(stream);

#ifdef linux
	checkCudaErrors(hipMemcpyFromSymbol(&h_callbackForwardNormZ, HIP_SYMBOL(d_callbackForwardNormZ), sizeof(h_callbackForwardNormZ)));
	checkCudaErrors(hipMemcpyFromSymbol(&h_callbackForwardNormD, HIP_SYMBOL(d_callbackForwardNormD), sizeof(h_callbackForwardNormD)));
	checkCudaErrors(hipMemcpyFromSymbol(&h_callbackInverseNormZ, HIP_SYMBOL(d_callbackInverseNormZ), sizeof(h_callbackInverseNormZ)));
	checkCudaErrors(hipMemcpyFromSymbol(&h_callbackInverseNormD, HIP_SYMBOL(d_callbackInverseNormD), sizeof(h_callbackInverseNormD)));
#endif

	callbackData = new double[2];

	callbackData[0] = L;
	callbackData[1] = N;
}

cuFFT::cuFFT(const int _dim, const int *_n, const int _BATCH, hipStream_t _stream) : dim(_dim), BATCH(_BATCH), stream(_stream)
{
	throw;
	n = new int[dim];
	for (size_t i = 0; i < dim; i++)
		n[i] = _n[i];

	int NX, NY, NZ;
	L = 1;

	setStream(stream);

	switch (dim)
	{
	case 1:
		/*NX = n[0];
		N = NX;

		if (hipfftPlan1d(&planZ2Z, NX, HIPFFT_Z2Z, BATCH) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			return;
		}
		if (hipfftPlan1d(&planD2Z, NX, HIPFFT_D2Z, BATCH) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			return;
		}
		if (hipfftPlan1d(&planZ2D, NX, HIPFFT_Z2D, BATCH) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			return;
		}
		break;*/
		throw;
	case 3:
		NX = n[0];
		NY = n[1];
		NZ = n[2];
		N = NX * NY * NZ;

		if (hipfftPlan3d(&planZ2ZF, NX, NY, NZ, HIPFFT_Z2Z) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			return;
		}
		if (hipfftPlan3d(&planD2Z, NX, NY, NZ, HIPFFT_D2Z) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			return;
		}
		if (hipfftPlan3d(&planZ2D, NX, NY, NZ, HIPFFT_Z2D) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			return;
		}

		break;
	
	default:
		throw;
	}
}
cuFFT::~cuFFT()
{
	checkCudaErrors(hipfftDestroy(planZ2ZF));
	checkCudaErrors(hipfftDestroy(planZ2ZI));
	checkCudaErrors(hipfftDestroy(planD2Z));
	checkCudaErrors(hipfftDestroy(planZ2D));
	delete[] n;
	delete[] callbackData;
}
void cuFFT::reset(const int _dim, const int *_n, double _L, const int _BATCH, hipStream_t _stream)
{
	dim = _dim;
	delete[] n;
	n = new int[dim];
	N = 1;
	for (size_t i = 0; i < dim; i++) {
		n[i] = _n[i];
		N *= n[i];
	}
	
	BATCH = _BATCH;
	L = _L;

	callbackData[0] = L;
	callbackData[1] = N;

	checkCudaErrors(hipfftDestroy(planZ2ZF));
	checkCudaErrors(hipfftDestroy(planZ2ZI));
	checkCudaErrors(hipfftDestroy(planD2Z));
	checkCudaErrors(hipfftDestroy(planZ2D));

	switch (dim)
	{
	case 1:
		/*NX = n[0];
		N = NX;

		if (hipfftPlan1d(&planZ2Z, NX, HIPFFT_Z2Z, BATCH) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			throw;
		}
		if (hipfftPlan1d(&planD2Z, NX, HIPFFT_D2Z, BATCH) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			throw;
		}
		if (hipfftPlan1d(&planZ2D, NX, HIPFFT_Z2D, BATCH) != HIPFFT_SUCCESS) {
			fprintf(stderr, "CUFFT error: Plan creation failed");
			throw;
		}
		break;*/
		throw;

	case 3:
		size_t workSize;
		
		checkCudaErrors(hipfftCreate(&planZ2ZF));
		checkCudaErrors(hipfftMakePlan3d(planZ2ZF, n[0], n[1], n[2], HIPFFT_Z2Z, &workSize));

		checkCudaErrors(hipfftCreate(&planZ2ZI));
		checkCudaErrors(hipfftMakePlan3d(planZ2ZI, n[0], n[1], n[2], HIPFFT_Z2Z, &workSize));

		checkCudaErrors(hipfftCreate(&planD2Z));
		checkCudaErrors(hipfftMakePlan3d(planD2Z, n[0], n[1], n[2], HIPFFT_D2Z, &workSize));

		checkCudaErrors(hipfftCreate(&planZ2D));
		checkCudaErrors(hipfftMakePlan3d(planZ2D, n[0], n[1], n[2], HIPFFT_Z2D, &workSize));

#ifdef linux
		checkCudaErrors(hipfftXtSetCallback(planZ2ZF, (void**)&h_callbackForwardNormZ, HIPFFT_CB_ST_COMPLEX_DOUBLE, (void**)callbackData));
		checkCudaErrors(hipfftXtSetCallback(planZ2ZI, (void**)&h_callbackInverseNormZ, HIPFFT_CB_ST_COMPLEX_DOUBLE, (void**)callbackData));
		checkCudaErrors(hipfftXtSetCallback(planD2Z, (void**)&h_callbackForwardNormZ, HIPFFT_CB_ST_COMPLEX_DOUBLE, (void**)callbackData));
		checkCudaErrors(hipfftXtSetCallback(planZ2D, (void**)&h_callbackInverseNormD, HIPFFT_CB_ST_REAL_DOUBLE, (void**)callbackData));
#endif

		//if (hipfftPlan3d(&planZ2Z, NX, NY, NZ, HIPFFT_Z2Z) != HIPFFT_SUCCESS) {
		//	fprintf(stderr, "CUFFT error: Plan creation failed");
		//	throw;
		//}
		//if (hipfftPlan3d(&planD2Z, NX, NY, NZ, HIPFFT_D2Z) != HIPFFT_SUCCESS) {
		//	fprintf(stderr, "CUFFT error: Plan creation failed");
		//	throw;
		//}
		//if (hipfftPlan3d(&planZ2D, NX, NY, NZ, HIPFFT_Z2D) != HIPFFT_SUCCESS) {
		//	fprintf(stderr, "CUFFT error: Plan creation failed");
		//	throw;
		//}

		break;

	default:
		throw;
	}

	stream = _stream;
	setStream(stream);
	std::cout << "55" << std::endl;
}